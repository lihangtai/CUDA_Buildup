inline hipError_t cudaMallocMapped( void** ptr, size_t size, bool clear=true )
{
	void* cpu = NULL;
	void* gpu = NULL;

	if( !ptr || size == 0 )
		return hipErrorInvalidValue;

	//CUDA_ASSERT(hipSetDeviceFlags(hipDeviceMapHost));

    CUDA_ASSERT(hipHostAlloc(&cpu, size, hipHostMallocMapped));
    CUDA_ASSERT(hipHostGetDevicePointer(&gpu, cpu, 0));

    if( cpu != gpu )
    {
        LogError(LOG_CUDA "cudaMallocMapped() - addresses of CPU and GPU pointers don't match (CPU=%p GPU=%p)\n", cpu, gpu);
        return hipErrorInvalidDevicePointer;
    }
    
    if( clear )
	    memset(cpu, 0, size);

    *ptr = cpu;
	return hipSuccess;
}
